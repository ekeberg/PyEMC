#include "hip/hip_runtime.h"

extern "C" __global__ void kernel_normalize_slices(float *const slices,
						   const float *responsabilities,
						   const int number_of_pixels,
						   const int number_of_patterns)
{
  __shared__ float normalization_factor_cache[NTHREADS];
  const int index_rotation = blockIdx.x;

  normalization_factor_cache[threadIdx.x] = 0.;
  for (int index_pattern = threadIdx.x;
       index_pattern < number_of_patterns;
       index_pattern += blockDim.x) {
    float this_resp = responsabilities[index_rotation*number_of_patterns + index_pattern];
    normalization_factor_cache[threadIdx.x] += this_resp;
  }
  inblock_reduce(normalization_factor_cache);
  float normalization_factor;
  if (normalization_factor_cache[0] > -10.) {
    normalization_factor = 1./normalization_factor_cache[0];
    //normalization_factor = normalization_factor_cache[0];
  } else {
    normalization_factor = 0.;
    //normalization_factor = normalization_factor_cache[0];
  }
  for (int index_pixel = threadIdx.x; index_pixel < number_of_pixels; index_pixel += blockDim.x) {
    slices[index_rotation*number_of_pixels + index_pixel] *= normalization_factor;
    //slices[index_rotation*number_of_pixels + index_pixel] *= 1./normalization_factor;
  }
}

extern "C" __global__ void kernel_update_slices(float *const slices,
						const int *const patterns,
						const int number_of_patterns,
						const int number_of_pixels,
						const float *const responsabilities)
{
  const int index_rotation = blockIdx.x;
  float sum;
  float weight;
  for (int pixel_index = threadIdx.x;
       pixel_index < number_of_pixels;
       pixel_index += blockDim.x) {
    sum = 0.;
    weight = 0.;
    for (int pattern_index = 0;
	 pattern_index < number_of_patterns;
	 pattern_index++) {
      if (patterns[pattern_index*number_of_pixels + pixel_index] >= 0.) {
	sum += (patterns[pattern_index*number_of_pixels + pixel_index] *
		responsabilities[index_rotation*number_of_patterns + pattern_index]);
	weight += responsabilities[index_rotation*number_of_patterns + pattern_index];
      }
    }
    if (weight > 0.) {
      slices[index_rotation*number_of_pixels + pixel_index] = sum / weight;
    } else {
      slices[index_rotation*number_of_pixels + pixel_index] = -1.;
    }
  }
}



/* extern "C" __global__ void kernel_update_slices(float *const slices, */
/* 						const int *const patterns, */
/* 						const int number_of_patterns, */
/* 						const int number_of_pixels, */
/* 						const float *const responsabilities) */
/* { */
/*   const int index_rotation = blockIdx.x; */

/*   for (int pattern_index = threadIdx.x; pattern_index < number_of_patterns; pattern_index += blockDim.x) { */
/*     for (int pixel_index = 0; pixel_index < number_of_pixels; pixel_index++) { */
/*       slices[index_rotation*number_of_pixels + pixel_index] += (patterns[pattern_index*number_of_pixels + pixel_index] * */
/* 								responsabilities[index_rotation*number_of_patterns + pattern_index]); */
/*     } */
/*   } */
	
  /* float sum; */
  /* float weight; */
  /* for (int pixel_index = threadIdx.x; */
  /*      pixel_index < number_of_pixels; */
  /*      pixel_index += blockDim.x) { */
  /*   sum = 0.; */
  /*   weight = 0.; */
  /*   for (int pattern_index = 0; */
  /* 	 pattern_index < number_of_patterns; */
  /* 	 pattern_index++) { */
  /*     if (patterns[pattern_index*number_of_pixels + pixel_index] >= 0.) { */
  /* 	sum += (patterns[pattern_index*number_of_pixels + pixel_index] * */
  /* 		responsabilities[index_rotation*number_of_patterns + pattern_index]); */
  /* 	weight += responsabilities[index_rotation*number_of_patterns + pattern_index]; */
  /*     } */
  /*   } */
  /*   if (weight > 0.) { */
  /*     slices[index_rotation*number_of_pixels + pixel_index] = sum / weight; */
  /*   } else { */
  /*     slices[index_rotation*number_of_pixels + pixel_index] = -1.; */
  /*   } */
  /* } */
/* } */



extern "C" __global__ void kernel_update_slices_scaling(float *const slices,
							const int *const patterns,
							const int number_of_patterns,
							const int number_of_pixels,
							const float *const responsabilities,
							const float *const scaling)
{
  const int index_rotation = blockIdx.x;
  float sum;
  float weight;
  for (int pixel_index = threadIdx.x; pixel_index < number_of_pixels; pixel_index += blockDim.x) {
    sum = 0.;
    weight = 0.;
    for (int pattern_index = 0;
	 pattern_index < number_of_patterns;
	 pattern_index += 1) {
      if (patterns[pattern_index*number_of_pixels + pixel_index] >= 0.) {
	sum += (patterns[pattern_index*number_of_pixels + pixel_index] *
		scaling[index_rotation*number_of_patterns + pattern_index] *
		responsabilities[index_rotation*number_of_patterns + pattern_index]);
	weight += responsabilities[index_rotation*number_of_patterns + pattern_index];
      }
    }
    if (weight > 0.) {
      slices[index_rotation*number_of_pixels + pixel_index] = sum / weight;
    } else {
      slices[index_rotation*number_of_pixels + pixel_index] = -1.;
    }
  }
}


extern "C" __global__ void kernel_update_slices_per_pattern_scaling(float *const slices,
								    const int *const patterns,
								    const int number_of_patterns,
								    const int number_of_pixels,
								    const float *const responsabilities,
								    const float *const scaling)
{
  const int index_rotation = blockIdx.x;
  float sum;
  float weight;
  for (int pixel_index = threadIdx.x; pixel_index < number_of_pixels; pixel_index += blockDim.x) {
    sum = 0.;
    weight = 0.;
    for (int pattern_index = 0; pattern_index < number_of_patterns; pattern_index++) {
      if (patterns[pattern_index*number_of_pixels + pixel_index] >= 0.) {
	sum += (patterns[pattern_index*number_of_pixels + pixel_index] *
		scaling[pattern_index] *
		responsabilities[index_rotation*number_of_patterns + pattern_index]);
	weight += responsabilities[index_rotation*number_of_patterns + pattern_index];
      }
    }
    if (weight > 0.) {
      slices[index_rotation*number_of_pixels + pixel_index] = sum / weight;
    } else {
      slices[index_rotation*number_of_pixels + pixel_index] = -1.;
    }
  }
}


/* This can't handle masks att the moment. Need to think about how to handle masked out data in the sparse implemepntation
 */
extern "C" __global__ void kernel_update_slices_sparse(float *const slices,
						       const int number_of_pixels,
						       const int *const pattern_start_indices,
						       const int *const pattern_indices,
						       const int *const pattern_values,
						       const int number_of_patterns,
						       const float *const responsabilities,
						       const float resp_threshold)
{
  //const int number_of_rotations = gridDim.x;
  const int index_rotation = blockIdx.x;
  
  int index_pixel;
  
  for (int index_pixel = threadIdx.x; index_pixel < number_of_pixels; index_pixel += blockDim.x) {
    slices[index_rotation*number_of_pixels + index_pixel] = 0.0;
  }
  __syncthreads();
  for (int index_pattern = 0; index_pattern < number_of_patterns; index_pattern++) {
    float this_resp = responsabilities[index_rotation*number_of_patterns + index_pattern];
    for (int value_index = pattern_start_indices[index_pattern] + threadIdx.x;
    	 value_index < pattern_start_indices[index_pattern+1];
    	 value_index += blockDim.x) {
      index_pixel = pattern_indices[value_index];
      if (this_resp > resp_threshold) {
	atomicAdd(&slices[index_rotation*number_of_pixels + index_pixel],
		  pattern_values[value_index] * this_resp);
      }
    }
  }
}


extern "C" __global__ void kernel_update_slices_sparse_scaling(float *const slices,
							       const int number_of_pixels,
							       const int *const pattern_start_indices,
							       const int *const pattern_indices,
							       const int *const pattern_values,
							       const int number_of_patterns,
							       const float *const responsabilities,
							       const float resp_threshold,
							       const float *const scaling)
{
  //const int number_of_rotations = gridDim.x;
  const int index_rotation = blockIdx.x;
  //const int index_pattern = blockIdx.x;

  int index_pixel;

  for (int index_pixel = threadIdx.x; index_pixel < number_of_pixels; index_pixel += blockDim.x) {
    slices[index_rotation*number_of_pixels + index_pixel] = 0.;
  }
  __syncthreads();
  for (int index_pattern = 0; index_pattern < number_of_patterns; index_pattern += 1) {
  /* int index_pattern = blockIdx.x; */
    float this_resp = responsabilities[index_rotation*number_of_patterns + index_pattern];
    float this_scaling = scaling[index_rotation*number_of_patterns + index_pattern];
    if (this_resp > resp_threshold) {
      for (int value_index = pattern_start_indices[index_pattern]+threadIdx.x;
	   value_index < pattern_start_indices[index_pattern+1];
	   value_index += blockDim.x) {
	index_pixel = pattern_indices[value_index];
	atomicAdd(&slices[index_rotation*number_of_pixels + index_pixel],
		  pattern_values[value_index] * this_scaling * this_resp);
      }
    }
  }
  /* float pixel_sum; */
  /* for (int pixel_index = threadIdx.x; pixel_index < number_of_pixels; pixel_index += blockDim.x) { */
  /*   for (int index_pattern = 0; index_pattern < number_of_patterns; index_pattern += 1) { */
  /*     float this_resp = responsabilities[index_rotation*number_of_patterns + index_pattern]; */
  /*     float this_scaling = scaling[index_rotation*number_of_patterns + index_pattern]; */
  /*     pixel_sum = 0.; */
  /*     for (int sparse_index = pattern_start_indices[index_pattern]; */
  /* 	   sparse_index < pattern_start_indices[index_pattern+1]; */
  /* 	   sparse_index += 1) { */
  /* 	if (pattern_indices[sparse_index] == pixel_index) { */
  /* 	  pixel_sum += pattern_values[sparse_index]*this_scaling*this_resp; */
  /* 	} */
  /*     } */
  /*   } */
  /*   slices[index_rotation*number_of_pixels + index_pixel] = pixel_sum; */
  /* } */
}

extern "C" __global__ void kernel_update_slices_sparse_per_pattern_scaling(float *const slices,
									   const int number_of_pixels,
									   const int *const pattern_start_indices,
									   const int *const pattern_indices,
									   const int *const pattern_values,
									   const int number_of_patterns,
									   const float *const responsabilities,
									   const float *const scaling)
{
  __shared__ float normalization_factor_cache[NTHREADS];
  //const int number_of_rotations = gridDim.x;
  const int index_rotation = blockIdx.x;

  int index_pixel;

  for (int index_pixel = threadIdx.x; index_pixel < number_of_pixels; index_pixel += blockDim.x) {
    slices[index_rotation*number_of_pixels + index_pixel] = 0.;
  }
  __syncthreads();

  /* for (int index_pattern = threadIdx.x; index_pattern < number_of_patterns; index_pattern += blockDim.x) { */
  /*   for (int value_index = pattern_start_indices[index_pattern]; value_index < pattern_start_indices[index_pattern+1]; value_index += 1) { */
  /*     index_pixel = pattern_indices[value_index]; */
  /*     atomicAdd(&slices[index_rotation*number_of_pixels + index_pixel], */
  /* 		pattern_values[value_index] * scaling[index_pattern] * */
  /* 		responsabilities[index_rotation*number_of_patterns + index_pattern]); */
  /*   } */
  /* } */

  for (int index_pattern = 0; index_pattern < number_of_patterns; index_pattern++) {
    for (int value_index = pattern_start_indices[index_pattern] + threadIdx.x;
	 value_index < pattern_start_indices[index_pattern+1];
	 value_index += blockDim.x) {
      index_pixel = pattern_indices[value_index];
      slices[index_rotation*number_of_pixels + index_pixel] += (pattern_values[value_index] *
								scaling[index_pattern] *
								responsabilities[index_rotation*number_of_patterns + index_pattern]);
    }
  }

  
  normalization_factor_cache[threadIdx.x] = 0.;
  for (int index_pattern = threadIdx.x; index_pattern < number_of_patterns; index_pattern += blockDim.x) {
    normalization_factor_cache[threadIdx.x] += responsabilities[index_rotation*number_of_patterns + index_pattern];
  }
  inblock_reduce(normalization_factor_cache);
  float normalization_factor = normalization_factor_cache[0];
  for (int index_pixel = threadIdx.x; index_pixel < number_of_pixels; index_pixel += blockDim.x) {
    slices[index_rotation*number_of_pixels + index_pixel] *= 1./normalization_factor;
  }
}
